#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "kernel_args.h"  // for intellisense
#include "score.h"

/**
 * Kernel to compute dot product of a single query vector relative to a batch of vectors.
 */
__global__ void multiplyAndSum(
    const float* batch,
    const float* query,
    float* dotResults,
    float* normResults,
    size_t batchSize,
    size_t vectorSize
) {
    for (size_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < batchSize * vectorSize; idx += blockDim.x * gridDim.x) {
        size_t batchIdx = idx / vectorSize;
        size_t vectorIdx = idx % vectorSize;
        atomicAdd(&dotResults[batchIdx], batch[idx] * query[vectorIdx]);
        atomicAdd(&normResults[batchIdx], batch[idx] * batch[idx]);
    }
}

__global__ void normalize(
    float* dotResults,
    const float* normResults,
    float queryNorm,
    size_t batchSize
) {
    for (size_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < batchSize; idx += blockDim.x * gridDim.x) {
        dotResults[idx] /= (sqrtf(normResults[idx]) * queryNorm);
    }
}


void cudaCosineSimilarity(
    const float* batch,
    const float* query,
    float* results,
    size_t batchSize,
    size_t vectorSize
) {
    float queryNorm = norm(query, vectorSize);

    // allocate cuda memory
    float* cudaBatch;
    float* cudaQuery;
    float* cudaDotResults;
    float* cudaNormResults;
    size_t batchMemSize = batchSize * sizeof(float);
    size_t queryMemSize = vectorSize * sizeof(float);
    size_t totalMemSize = batchSize * vectorSize * sizeof(float);
    hipMalloc(&cudaBatch, totalMemSize);
    hipMalloc(&cudaQuery, queryMemSize);
    hipMalloc(&cudaDotResults, batchMemSize);
    hipMalloc(&cudaNormResults, batchMemSize);
    hipMemcpy(cudaBatch, batch, totalMemSize, hipMemcpyHostToDevice);
    hipMemcpy(cudaQuery, query, queryMemSize, hipMemcpyHostToDevice);
    hipMemset(cudaDotResults, 0, batchMemSize);
    hipMemset(cudaNormResults, 0, batchMemSize);

    // run kernel
    int threads = 256;
    int blocks = std::min(65535, ((int)(batchSize * vectorSize) + threads - 1) / threads);
    multiplyAndSum KERNEL_ARGS2(blocks, threads) (cudaBatch, cudaQuery, cudaDotResults, cudaNormResults, batchSize, vectorSize);
    hipDeviceSynchronize();

    blocks = std::min(65535, ((int)batchSize + threads - 1) / threads);
    normalize KERNEL_ARGS2(blocks, threads) (cudaDotResults, cudaNormResults, queryNorm, batchSize);
    hipDeviceSynchronize();

    // copy results to cpu and free cuda memory
    hipMemcpy(results, cudaDotResults, batchMemSize, hipMemcpyDeviceToHost);
    hipFree(cudaBatch);
    hipFree(cudaQuery);
    hipFree(cudaDotResults);
    hipFree(cudaNormResults);
}
